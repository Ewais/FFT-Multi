#include "hip/hip_runtime.h"
#include "../../inlcude/Comp/CUDA_FFT_Comp.h"

__global__ void CUDA_ButterFly(hipFloatComplex* Spectrum_GPU,int Size)
{
    /** This function implements radix-2 butterfly (BF)
    * Executed by each thread in GPU
    * Inputs : Spectrum_GPU : Complex array in GPU memory
    *          Size : number of samples of the signal "N"
    * Outputs : saved in the same places in Spectrum_GPU
    */
	int Index = 2*Size*((int)((512*blockIdx.x + threadIdx.x)/Size))+((512*blockIdx.x + threadIdx.x)%Size);
	hipFloatComplex First = Spectrum_GPU[Index];
	hipFloatComplex Second = Spectrum_GPU[Index+Size];
	float Harmonic = -(2*Pi*Index)/(2*Size);
	hipFloatComplex Sin;
	Sin.x = cos(Harmonic);
	Sin.y = -sin(Harmonic);
	Spectrum_GPU[Index] = hipCaddf(First,hipCmulf(Second,Sin));
	Spectrum_GPU[Index+Size] = hipCsubf(First,hipCmulf(Second,Sin));
}

void CUDA_FFT_Comp(void)
{
    /** Implements composition of Signal in CUDA
    * uses radix-2 butterfly
    * Inputs:  None.
    * 	Input is global Array Decomposed
    * Output:  None.
    * 	Output is global array Spectrum
    */
	hipFloatComplex* Spectrum_GPU;
	hipMalloc(&Spectrum_GPU,sizeof(Decomposed));
	hipMemcpy(Spectrum_GPU,&Decomposed,sizeof(Decomposed),hipMemcpyHostToDevice);
	for(int Size = 1;Size <SIZE;Size*=2)
		//CUDA_ButterFly<<<Blocks,MaxThread>>>(Spectrum_GPU,Size);
		CUDA_ButterFly<<<1,32>>>(Spectrum_GPU,Size);
	hipMemcpy(&Spectrum,Spectrum_GPU,sizeof(Decomposed),hipMemcpyDeviceToHost);
	hipFree(Spectrum_GPU);
}
